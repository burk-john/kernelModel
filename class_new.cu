
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <stdio.h>

using namespace std;

//FC Layer using cuda
//Ni = size of input (I)
//Nn = size of output (O)
//B  = number of batches
//
//ONES  if true I and W (weights) are filled with 1s
//      else I and W are filled with random numbers (-0.5, 0.5)

#define Ni 4096     //25088
#define Nn 25088   //4096   //4096
#define B  256

#define Blocks 128

#define ONES true

float I_vals[B][Ni];
float W_vals[Ni][Nn];
float O_vals[B][Nn];

__global__
void fc(float I[][Ni], float W[][Nn], float O[][Nn]){
    float sum = 0;

    int n_start = blockIdx.x * (Nn / gridDim.x) + threadIdx.x;
    int n_end = n_start + Nn / gridDim.x / blockDim.x;

    for(int n = n_start; n < n_end; n++){
        for(int b = 0; b < B; b++){
            for(int i = 0; i < Ni; i++){
                sum += I[b][i] * W[i][n];
            }
            O[b][n] = sum;
            sum = 0;
        }
    }
}

int main() {
    float *I, *W, *O;

    hipMallocManaged(&I, B*Ni*sizeof(float));
    hipMallocManaged(&W, Ni*Nn*sizeof(float));
    hipMallocManaged(&O, B*Nn*sizeof(float));

    //Initialize I
    for(int b = 0; b < B; b++) {
        for(int i = 0; i < Ni; i++) {
            if(ONES) {
                I_vals[b][i] = 1;
            } else {
                I_vals[b][i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
            }
        }
    }

    //Initialize W
    for(int i = 0; i < Ni; i++){
        for(int n = 0; n < Nn; n++){
            if(ONES) {
                W_vals[i][n] = 1;
            } else {
                W_vals[i][n] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX) - 0.5f;
            }
        }
    }
    //Initialize O with zeros
    for(int b = 0; b < B; b++) {
        for(int n = 0; n < Nn; n++){
            O_vals[b][n] = 0;
        }
    }

    std::copy(&I_vals[0][0], &I_vals[0][0] + B*Ni, I);
    std::copy(&W_vals[0][0], &W_vals[0][0] + Ni*Nn, W);
    std::copy(&O_vals[0][0], &O_vals[0][0] + B*Nn, O);

    int threads = 0;
    if(Nn / Blocks > 1024){
        threads = 1024;
    } else {
        threads = Nn / Blocks;
    }

    fc<<<Blocks, threads>>>( reinterpret_cast<float (*)[Ni]>(I),
                    reinterpret_cast<float (*)[Nn]>(W),
                    reinterpret_cast<float (*)[Nn]>(O) );

    hipDeviceSynchronize();

    //Test when I and W initialized with 1s
    int err = 0;
    for(int b = 0; b < B; b++) {
        for(int n = 0; n < Nn; n++){
            if(ONES && (reinterpret_cast<float (*)[Nn]>(O)[b][n] != Ni)){
                err++;
            }
        }
    }

    if(ONES) {
        cout<<"Number of errors: "<<err<<endl;
    } else {
        cout<<"Error could not be calculated"<<endl;
    }

    hipFree(I);
    hipFree(W);
    hipFree(O);

    return 0;
}
